#include "hip/hip_runtime.h"
#include "caffe/util/math_functions.hpp"
namespace caffe {

template <typename Dtype>
__global__ void ClampData(int N, Dtype* data, Dtype low, Dtype high) {
  CUDA_KERNEL_LOOP(i, N) {
    data[i] = (data[i] > high) ? (high) : (data[i]);
    data[i] = (data[i] < low) ? (low) : (data[i]);
  }
}

template <typename Dtype>
void Clamp_data_gpu(int N, Dtype* data, Dtype low, Dtype high) {
  ClampData<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, data, low, high);
  CUDA_POST_KERNEL_CHECK;
}
template void Clamp_data_gpu<float>(int, float*, float, float);
template void Clamp_data_gpu<double>(int, double*, double, double);

}

